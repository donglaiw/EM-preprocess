#include "hip/hip_runtime.h"
/**********************************************************************************************************************
 * Name: TorchExtensionKernel.cpp
 * Author: Matin Raayai Ardakani
 * Email: raayai.matin@gmail.com
 * Contains the CUDA kernels written using Pytorch's Aten backend.
 * For function documentation, refer to the associated header file.
 **********************************************************************************************************************/
 #include "MedianFilter.h"


at::Tensor median_filter_cuda(const at::Tensor& imStack) {
    at::Tensor out = at::zeros_like(imStack[0]);
    const int32_t dimX = imStack.size(2), dimY = imStack.size(1), dimZ = imStack.size(0);
    const dim3 blockDim(BLOCK_DIM_LEN, BLOCK_DIM_LEN, 1);
    const dim3 gridDim((dimX / blockDim.x + ((dimX % blockDim.x) ? 1 : 0)),
            (dimY / blockDim.y + ((dimY % blockDim.y) ? 1 : 0)), 1);

    AT_DISPATCH_FLOATING_TYPES(imStack.type(), "median_filter_kernel", ([&] {
        median_filter_kernel<scalar_t><<<gridDim, blockDim>>>(
            imStack.data<scalar_t>(),
            out.data<scalar_t>(),
            dimX,
            dimY,
            dimZ);
      }));
    return out;
}

at::Tensor median_filter_cuda(const at::Tensor& imStack, const int radX, const int radY, const int radZ) {

    at::Tensor out = at::zeros_like(imStack);
    const int32_t dimX = imStack.size(2), dimY = imStack.size(1), dimZ = imStack.size(0);

    const dim3 blockDim(BLOCK_DIM_LEN, BLOCK_DIM_LEN, BLOCK_DIM_LEN);
    const dim3 gridDim(
            (dimX/blockDim.x + ((dimX%blockDim.x)?1:0)),
            (dimY/blockDim.y + ((dimY%blockDim.y)?1:0)),
            (dimZ/blockDim.z + ((dimZ%blockDim.z)?1:0)));

    AT_DISPATCH_FLOATING_TYPES(imStack.type(), "median_filter_kernel", ([&] {
        median_filter_kernel<scalar_t><<<gridDim, blockDim>>>(
                        imStack.data<scalar_t>(),
                        out.data<scalar_t>(),
                        dimX,
                        dimY,
                        dimZ,
                        radX,
                        radY,
                        radZ);
    }));
    return out;
}

template<typename scalar_t>
__global__
void median_filter_kernel(scalar_t* __restrict__ stackIn,
        scalar_t* __restrict__ imOut,
        int32_t dimX,
        int32_t dimY,
        int32_t dimZ) {
    auto get_1d_idx = [&] (int32_t x, int32_t y, int32_t z) {
        return clamp_mirror(z, 0, dimZ - 1) * dimY * dimX +
        clamp_mirror(y, 0, dimY - 1) * dimX + clamp_mirror(x, 0, dimX - 1);
    };

    const int32_t col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    const int32_t sht_idx = dimZ / 2;
	scalar_t windowVec[MAX_GPU_ARRAY_LEN] = {0.};
    int32_t vSize = 0;

    for (int32_t z = -dimZ; z <= dimZ; z++)
        windowVec[vSize++] = stackIn[get_1d_idx(col_idx, row_idx, sht_idx + z)];

    imOut[get_1d_idx(col_idx, row_idx, sht_idx)] = get_median_of_array(windowVec, vSize);
}


template<typename scalar_t>
__global__
void __median_3d(scalar_t* __restrict__ stackIn,
        scalar_t* __restrict__ stackOut,
        int32_t dimX,
        int32_t dimY,
        int32_t dimZ,
        int32_t radX,
        int32_t radY,
        int32_t radZ)
    {
    auto get_1d_idx = [&] (int32_t x, int32_t y, int32_t z) {
        return clamp_mirror(z, 0, dimZ - 1) * dimY * dimX + 
        clamp_mirror(y, 0, dimY - 1) * dimX + clamp_mirror(x, 0, dimX - 1);
    };

    const int32_t col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    const int32_t sht_idx = blockIdx.z * blockDim.z + threadIdx.z;

	scalar_t windowVec[MAX_GPU_ARRAY_LEN] = {0.};
    int32_t vSize = 0;

    for (int32_t z = -radZ; z <= radZ; z++)
    for (int32_t y = -radY; y <= radY; y++)
    for (int32_t x = -radX; x <= radX; x++)
        windowVec[vSize++] = stackIn[get_1d_idx(x + col_idx, y + row_idx, z + sht_idx)];

    stackOut[get_1d_idx(col_idx, row_idx, sht_idx)] = calculate_median(windowVec, vSize);
}

#include "hip/hip_runtime.h"
/**********************************************************************************************************************
 * Name: TorchExtensionKernel.cpp
 * Author: Matin Raayai Ardakani
 * Email: raayai.matin@gmail.com
 * Where the CUDA magic happens for the em_pre_cuda Python package.
 * Based on the code from Pytorch's tutorials: https://github.com/pytorch/extension-cpp
 **********************************************************************************************************************/
 #include "TorchExtensionKernel.h"

/**
 * Applies a median filter to the image stack with a window shape of [1, 1, 2 * radZ + 1] and returns the middle slice.
 * @param imStack input image stack as a ATen CUDA tensor with float data type.
 * @param radZ the z-radius of the median filter.
 * @return the middle slice of the output of the median filter as an ATen CUDA Tensor with float data type.
 */
at::Tensor cuda_3d_median(const at::Tensor& imStack) {
    at::Tensor imStackOut = at::zeros_like(imStack[0]);
    const int32_t dimX = imStack.size(2), dimY = imStack.size(1), dimZ = imStack.size(0);
    const dim3 blockDim(BLOCK_DIM_LEN, BLOCK_DIM_LEN);
    const dim3 gridDim((dimX/blockDim.x + ((dimX%blockDim.x)?1:0)), (dimY/blockDim.y + ((dimY%blockDim.y)?1:0)));

    AT_DISPATCH_FLOATING_TYPES(imStack.type(), "__median_3d", ([&] {
        __median_3d<scalar_t><<<gridDim, blockDim>>>(
            imStack.data<scalar_t>(),
            imStackOut.data<scalar_t>(),
            dimX,
            dimY,
            dimZ);
      }));
    return imStackOut;
}

/**
 * A getter used by the threads in each kernel to access a 3d slice stack.
 */
inline __device__ __host__ int clamp_mirror(int idx, int minIdx, int maxIdx)
{
    if(idx < minIdx) return (minIdx + (minIdx - idx));
    else if(idx > maxIdx) return (maxIdx - (idx - maxIdx));
    else return idx;
}

template<typename scalar_t>
__global__
void __median_3d(scalar_t* __restrict__ imStackIn, scalar_t* __restrict__ sliceOut, int32_t dimX, int32_t dimY,
    int32_t dimZ) {

    auto get_1d_idx = [&] (int32_t x, int32_t y, int32_t z) {
        return clamp_mirror(z, 0, dimZ - 1) * dimY * dimX +
        clamp_mirror(y, 0, dimY - 1) * dimX + clamp_mirror(x, 0, dimX - 1);
    };

    const int32_t col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t row_idx = blockIdx.y * blockDim.y + threadIdx.y;

	float windowVec[MAX_GPU_ARRAY_LEN] = {0.};
    int32_t vSize = 0;

    for (int32_t z = -dimZ; z <= dimZ; z++)
        windowVec[vSize++] = imStackIn[get_1d_idx(col_idx, row_idx, z)];

	for (int32_t i = 0; i < vSize; i++) {
		for (int32_t j = i + 1; j < vSize; j++) {
			if (windowVec[i] > windowVec[j]) {
				float tmp = windowVec[i];
				windowVec[i] = windowVec[j];
				windowVec[j] = tmp;
			}
		}
    }

    sliceOut[get_1d_idx(col_idx, row_idx, 0)] = windowVec[vSize/2];   //Set the output variables.
}

at::Tensor cuda_median_3d(const at::Tensor& imStack, const at::Tensor& filtRads) {

    at::Tensor imStackOut = at::zeros_like(imStack);
    const int32_t dimX = imStack.size(2), dimY = imStack.size(1), dimZ = imStack.size(0);
    auto f_copy = filtRads;
    //TODO: Make this accept all types.
    auto fa = f_copy.accessor<float, 1>();
    const int32_t radX = static_cast<int32_t>(fa[2]);
    const int32_t radY = static_cast<int32_t>(fa[1]);
    const int32_t radZ = static_cast<int32_t>(fa[0]);

    const dim3 blockDim(BLOCK_DIM_LEN, BLOCK_DIM_LEN, BLOCK_DIM_LEN);
    const dim3 gridDim(
        (dimX/blockDim.x + ((dimX%blockDim.x)?1:0)),
        (dimY/blockDim.y + ((dimY%blockDim.y)?1:0)),
        (dimZ/blockDim.z + ((dimZ%blockDim.z)?1:0)));

    AT_DISPATCH_FLOATING_TYPES(imStack.type(), "__median_3d", ([&] {
        __median_3d<scalar_t><<<gridDim, blockDim>>>(
            imStack.data<scalar_t>(),
            imStackOut.data<scalar_t>(),
            dimX,
            dimY,
            dimZ,
            radX,
            radY,
            radZ);
      }));
    return imStackOut;
}

template<typename scalar_t>
__global__
void __median_3d(scalar_t* __restrict__ imStackIn, 
    scalar_t* __restrict__ imStackOut,
    int32_t dimX,
    int32_t dimY,
    int32_t dimZ,
    int32_t radX,
    int32_t radY,
    int32_t radZ) 
    {

    auto get_1d_idx = [&] (int32_t x, int32_t y, int32_t z) {
        return clamp_mirror(z, 0, dimZ - 1) * dimY * dimX + 
        clamp_mirror(y, 0, dimY - 1) * dimX + clamp_mirror(x, 0, dimX - 1);
    };

    const int32_t col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    const int32_t sht_idx = blockIdx.z * blockDim.z + threadIdx.z;

	scalar_t windowVec[MAX_GPU_ARRAY_LEN] = {0.};
    int32_t vSize = 0;

    for (int32_t z = -radZ; z <= radZ; z++)
    for (int32_t y = -radY; y <= radY; y++)
    for (int32_t x = -radX; x <= radX; x++)
        windowVec[vSize++] = imStackIn[get_1d_idx(x + col_idx, y + row_idx, z + sht_idx)];
        
	for (int32_t i = 0; i < vSize; i++) {
		for (int32_t j = i + 1; j < vSize; j++) {
			if (windowVec[i] > windowVec[j]) { 
				scalar_t tmp = windowVec[i];
				windowVec[i] = windowVec[j];
				windowVec[j] = tmp;
			}
		}
    }

    imStackOut[get_1d_idx(col_idx, row_idx, sht_idx)] = windowVec[vSize/2];   //Set the output variables.
}
